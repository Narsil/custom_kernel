#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <THC/THCAtomics.cuh>
#include <ATen/ATen.h>
#include <torch/torch.h>
#include <vector>

#include <optional>

/**
* Friendly reminder of how multithreading works in CUDA: https://developer.nvidia.com/blog/even-easier-introduction-cuda
* Check example at https://github.com/thomasw21/LinearTransformers/blob/main/model/attention/fast_weight/fast_weight_cuda.cu
**/

// Available in pytorch main
//#define DISPATCH_CASE_FLOATING_TYPES(...) \
//  at::AT_DISPATCH_CASE(at::ScalarType::Double, __VA_ARGS__) \
//  at::AT_DISPATCH_CASE(at::ScalarType::Float, __VA_ARGS__) \
//  at::AT_DISPATCH_CASE(at::ScalarType::Half, __VA_ARGS__) \
//  at::AT_DISPATCH_CASE(at::ScalarType::BFloat16, __VA_ARGS__) \

/*
* Forward passes
*/

/**
* cast to fp32 if in fp16 + mask + softmax computation in fp32 + cast back to original dtype
**/
template<typename attention_scores_scalar>
__global__ void forward_masked_softmax_kernel(
    const torch::PackedTensorAccessor32<attention_scores_scalar, 3, torch::RestrictPtrTraits> attention_scores, // [B, N, D]
    const torch::PackedTensorAccessor32<bool, 3, torch::RestrictPtrTraits> mask, // [B, N, D]
    torch::PackedTensorAccessor32<attention_scores_scalar, 3, torch::RestrictPtrTraits> result // [B, N, D]
) {
    const int batch_id = blockIdx.x;
    const int q_length_id = blockIdx.y;
    const int kv_length_id = threadIdx.x;

    // We need 2 float storage, one for max computation, the other for normalizing exponential
    __shared__ float temp_storage[2];
    if (kv_length_id == 0) {
        temp_storage[0] = -std::numeric_limits<float>::infinity();
        temp_storage[1] = 0;
    }
    __syncthreads();

    // Compute mask
    float elt;
    const auto mask_elt = mask[batch_id][q_length_id][kv_length_id];
    if (mask_elt == 1) {
        elt = -std::numeric_limits<float>::infinity();
    } else {
        elt = attention_scores[batch_id][q_length_id][kv_length_id];
        gpuAtomicMax(&temp_storage[0], elt);
    }

    // Compute max
    // TODO @thomasw21 get a MUCH faster sum mechanism in parallel?
    __syncthreads();

    // Compute exp(elt - max) masked
    float exponential;
    if (mask_elt == 1) {
        exponential = 0;
    } else {
        exponential = std::exp(elt - temp_storage[0]);
        gpuAtomicAdd(&temp_storage[1], exponential);
    }

    // Compute sum of exponential
    __syncthreads();

    // Compute softmax
    result[batch_id][q_length_id][kv_length_id] = static_cast<attention_scores_scalar>(exponential / temp_storage[1]);
}

extern "C" {
void fused_forward(
    const at::Tensor fused_qkv,
    at::Tensor layer_past_key,
    at::Tensor layer_past_value,
    const at::Tensor alibi,
    const at::Tensor attention_mask,
    const float beta,
    const float inv_norm_factor,
    const int num_heads,
    const bool use_cache,
    at::Tensor context_layer,
    at::Tensor attention_probs
) {
    std::cout<<"OKAY"<<std::endl;
    const auto batch_size = fused_qkv.size(0);
    const auto q_length = fused_qkv.size(1);
    const auto three_times_hidden_size = fused_qkv.size(2);
    const auto head_dim = three_times_hidden_size / (3 * num_heads);
    const auto batch_size_times_num_heads = batch_size * num_heads;

    std::cout<<"Batch size"<<batch_size<<std::endl;
    std::cout<<"Q size"<<q_length<<std::endl;
    std::cout<<"3 * H "<<three_times_hidden_size<<std::endl;
    std::cout<<"HD"<<head_dim<<std::endl;
    std::cout<<"BNH"<<batch_size_times_num_heads<<std::endl;

    // `split_heads`
    // std::cout<<"fused size"<<fused_qkv.sizes()<<std::endl;
    std::cout<<"fused view"<<batch_size << q_length <<num_heads << 3 * head_dim<<std::endl;
    const auto fused_qkv_view = fused_qkv.view({batch_size, q_length, num_heads, 3 * head_dim});
    std::cout<<"1"<<std::endl;
    const auto tensor_list = fused_qkv_view.split(head_dim, -1);
    std::cout<<"2"<<std::endl;
    const auto query_layer = tensor_list[0].transpose(1, 2).reshape({batch_size_times_num_heads, q_length, head_dim});
    std::cout<<"3"<<std::endl;
    auto key_layer = tensor_list[1].permute({0, 2, 3, 1}).reshape({batch_size_times_num_heads, head_dim, q_length});
    std::cout<<"4"<<std::endl;
    auto value_layer = tensor_list[2].transpose(1, 2).reshape({batch_size_times_num_heads, q_length, head_dim});
    std::cout<<"5"<<std::endl;
    std::cout<<"Reshapes done"<<std::endl;

    layer_past_key = at::cat({layer_past_key, key_layer}, 2);
    layer_past_value = at::cat({layer_past_value, value_layer}, 1);
    std::cout<<"Cat done"<<std::endl;

    auto attention_scores = alibi.baddbmm(query_layer, layer_past_key, beta, inv_norm_factor);
    std::cout<<"baddmm done"<<std::endl;

    if (true) {
        attention_probs = at::empty_like(attention_scores);
        const auto kv_length = key_layer.size(2);
        // TODO @thomasw21: Check that input are both in the correct device + contiguous

        // TODO @thomas21: change by to this as it's cleaner when pytorch 1.13 comes out
        // DISPATCH_CASE_FLOATING_TYPES(key_layer.scalar_type(), "masked_softmax", [&] {
	    std::cout<<"kernel launch"<<std::endl;
        AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, layer_past_key.scalar_type(), "masked_softmax", [&] {
            // TODO @thomasw21 I think this is necessary if you want to support all kinds of gpus.
            // const uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];

            /*
            * Understanding how GPUs work: https://developer.nvidia.com/blog/cuda-refresher-cuda-programming-model/
            * A100 specifications: https://images.nvidia.com/aem-dam/en-zz/Solutions/data-center/nvidia-ampere-architecture-whitepaper.pdf
            *  - SMs: 108
            *  - TPCs: 56 (What's that?)
            *  - Memory size: 40 GB
            *  - L2 Cache size: 40960 KB (shared across all SMs)
            *  - L1/Shared memory size: 192 KB (shared across all threads within a SM)
            *  - Max Threads / SM: 2048
            *  - Max Thread Blocks / SM: 32
            */

            /*
            * We should split [batch_size_times_num_heads, q_length] in seperate blocks and [kv_length] a single block
            * with multiple threads as we need to `sync_threads` to run exponential sum.
            */
            // TODO @thomasw21: Figure out how much I need exactly
	    std::cout<<"In here"<<std::endl;
            dim3 gridDim(batch_size_times_num_heads, q_length); // Number of blocks that run
            dim3 blockDim(kv_length); // Number of threads that run per block
            // TODO @thomasw21: Figure out how much I need
            const int shared_mem_forward = 2 * sizeof(float);

            // 192 * 2 ** 10
            const auto MAX_L1_MEMORY = 196608;
            const auto MAX_SMs = 108;
            const auto MAX_THREADS_PER_SM = 2048;
            TORCH_CHECK(batch_size_times_num_heads * q_length < MAX_L1_MEMORY, "Shared memory exceeds 192KB limitation.");
            // TORCH_CHECK(gridDim.x * gridDim.y * gridDim.z < MAX_SMs, "A100s only have 108 SMs. Raising as require blocks is bigger.");
            TORCH_CHECK(blockDim.x * blockDim.y * blockDim.z < MAX_THREADS_PER_SM, "A100s only have 2048 threads per block. Raising as require requested threads is higher.");

	    std::cout<<"Just before launch"<<std::endl;
            forward_masked_softmax_kernel<<<gridDim, blockDim, shared_mem_forward>>>(
                attention_scores.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                attention_mask.packed_accessor32<bool, 3, torch::RestrictPtrTraits>(),
                attention_probs.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>()
            );
        });
    } else {
	    std::cout<<"Not a kernel"<<std::endl;
        auto input_dtype = attention_scores.scalar_type();
	    std::cout<<"input dtype"<<input_dtype<<std::endl;
        if (input_dtype == at::ScalarType::Float) {
            attention_scores = attention_scores.to(at::ScalarType::Float);
        };
    std::cout<<"After cast"<<input_dtype<<std::endl;
    std::cout<<"Attention scores"<<attention_scores.sizes()<<std::endl;
    std::cout<<"Attention_mask"<<attention_mask.sizes()<<std::endl;
        // TODO @thomasw21 Figure out how to get minimum value
        auto attn_weights = attention_scores.masked_fill_(attention_mask, -1e34);
    std::cout<<"masked fill"<<input_dtype<<std::endl;
        attention_probs = attn_weights.softmax(-1, at::ScalarType::Float).to(input_dtype);
        std::cout<<"cast back"<<input_dtype<<std::endl;
    }
    std::cout<<"kernel launch done"<<std::endl;

    context_layer = attention_probs.bmm(layer_past_value);

    std::cout<<"bmm done"<<std::endl;
    // `_merge_heads`
    context_layer = context_layer.view({batch_size, num_heads, q_length, head_dim});
    context_layer = context_layer.permute({0, 2, 1, 3});
    context_layer = context_layer.reshape({batch_size, q_length, three_times_hidden_size / 3});
    std::cout<<"final reshape"<<std::endl;

}
}
